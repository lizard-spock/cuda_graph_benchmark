#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include "setup.h"

#define N 100000  // Adjusted size of vectors for 16 GB memory per GPU
#define NUM_GRAPHS 100
#define NUM_OPERATIONS 200

__global__ void vectorAdd(const float* A, const float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void _check_cuda(hipError_t err, std::string filename, int line) {
  if(err != hipSuccess) {
    throw std::runtime_error(
      "cuda no success at " + filename + ":" + write_with_ss(line));
  }
}
#define check_cuda(call) _check_cuda(call, __FILE__, __LINE__)

double flops(uint64_t ni, uint64_t nj, uint64_t nk, int nmm, float msec)
{
  double f = 1.0*(ni*nj*nk*uint64_t(nmm));
  double ret = f / double(msec);
  ret *= 1000.0;
  DOUT("in flops")
  DOUT(ret)
  return ret;
}

// struct event_loop_t {
//   event_loop_t(hipStream_t s, env_t& e, data_env_t& d)
//     : stream(s), env(e), data(d)
//   {}

//   void run(int n) {
//     while(n != 0) {
//       launch();
//       std::unique_lock lk(m_notify);
//       cv_notify.wait(lk);
//       n -= 1;
//     }
//   }

//   void launch() {
//     vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, N);

//     check_cuda(hipStreamAddCallback(
//       stream,
//       [](hipStream_t stream, hipError_t status, void* user_data) {
//         event_loop_t* self = reinterpret_cast<event_loop_t*>(user_data);
//         self->callback();
//       },
//       reinterpret_cast<void*>(this),
//       0));
//   };

//   void callback() {
//     {
//       std::unique_lock lk(m_notify);
//       // modify the shared state here (there isn't any)
//     }

//     cv_notify.notify_one();
//   }

//   hipStream_t stream;
//   env_t& env;
//   data_env_t& data;

//   std::mutex m_notify;
//   std::condition_variable cv_notify;
// };

/* This main function creates only one cuda graph and runs it */
int main_cudagraph01() {
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Initialize vectors
    std::vector<float> h_A(N, 1.0f);
    std::vector<float> h_B(N, 2.0f);
    hipMemcpy(d_A, h_A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipStream_t stream;
    hipStreamCreate(&stream);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    bool graphCreated = false;
    hipGraph_t graph;
    hipGraphExec_t instance;

    if (!graphCreated) {
        auto initstart = std::chrono::high_resolution_clock::now();
        hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
        for (int j = 0; j < NUM_OPERATIONS; ++j) {
            vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, N);
        }
        hipStreamEndCapture(stream, &graph);
        hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
        graphCreated = true;
        auto initend = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> elapsed = initend - initstart;
        std::cout << "Graph Create time: " << elapsed.count() << " milliseconds\n";
    }

    hipEvent_t start, stop;
    check_cuda(hipEventCreate(&start));
    check_cuda(hipEventCreate(&stop));
    check_cuda(hipEventRecord(start, stream));

    auto execstart = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < NUM_GRAPHS; ++i) {
        hipGraphLaunch(instance, stream);
    }
    hipStreamSynchronize(stream);
    auto execend = std::chrono::high_resolution_clock::now();
    
    check_cuda(hipEventRecord(stop, stream));
    check_cuda(hipEventSynchronize(stop));


    float msec = 0.0f;
    check_cuda(hipEventElapsedTime(&msec, start, stop));

    std::chrono::duration<double, std::milli> elapsed = execend - execstart;
    // std::cout << "Time for 100 CUDA Graphs with stream captures: " << elapsed.count() << " milliseconds\n";
    std::cout << "Time for 100 CUDA Graphs with hipEventRecord: " << msec << " milliseconds\n";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(stream);
    hipGraphDestroy(graph);
    hipGraphExecDestroy(instance);

    return 0;
}

int main_stream01() {
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Initialize vectors
    std::vector<float> h_A(N, 1.0f);
    std::vector<float> h_B(N, 2.0f);
    hipMemcpy(d_A, h_A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipStream_t stream;
    hipStreamCreate(&stream);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    hipEvent_t start, stop;
    check_cuda(hipEventCreate(&start));
    check_cuda(hipEventCreate(&stop));
    check_cuda(hipEventRecord(start, stream));
    
    for (int i = 0; i < NUM_GRAPHS; ++i) {
        for (int j = 0; j < NUM_OPERATIONS; ++j) {
            vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, N);
        }
    }
    hipStreamSynchronize(stream);
    
    check_cuda(hipEventRecord(stop, stream));
    check_cuda(hipEventSynchronize(stop));
    float msec = 0.0f;
    check_cuda(hipEventElapsedTime(&msec, start, stop));

    std::cout << "Time for 100 stream with hipEventRecord: " << msec << " milliseconds\n";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(stream);

    return 0;
}


int main(int argc, char** argv) {
    main_cudagraph01();
    main_stream01();
    return 0;
}