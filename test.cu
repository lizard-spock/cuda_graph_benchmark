#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    // Define vector size
    int N = 1000;
    size_t size = N * sizeof(float);

    // Allocate vectors on host
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate vectors on device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Create CUDA graph and stream
    hipGraph_t graph;
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    // Define execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel and capture in the graph
    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C);

    // End capture
    hipStreamEndCapture(stream, &graph);

    // Create executable graph
    hipGraphExec_t graphExec;
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    // Launch the executable graph
    hipGraphLaunch(graphExec, stream);

    // Synchronize stream
    hipStreamSynchronize(stream);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Validate result
    for (int i = 0; i < N; i++) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    std::cout << "Test PASSED" << std::endl;

    // Clean up
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    hipStreamDestroy(stream);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
