#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <chrono>

#define N 1000000  // Size of vectors
#define NUM_OPERATIONS 2000

__global__ void vectorAdd(const float* A, const float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Initialize vectors
    std::vector<float> h_A(N, 1.0f);
    std::vector<float> h_B(N, 2.0f);
    hipMemcpy(d_A, h_A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipStream_t stream;
    hipStreamCreate(&stream);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < NUM_OPERATIONS; ++i) {
        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, N);
    }
    hipStreamSynchronize(stream);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Time for 2000 operations with streams: " << elapsed.count() << " seconds\n";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(stream);

    return 0;
}
