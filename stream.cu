#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include "setup.h"


#define N 10000000  // Adjusted size of vectors for 16 GB memory per GPU
#define NUM_GRAPHS 100
#define NUM_OPERATIONS 200

__global__ void vectorAdd(const float* A, const float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void _check_cuda(hipError_t err, std::string filename, int line) {
  if(err != hipSuccess) {
    throw std::runtime_error(
      "cuda no success at " + filename + ":" + write_with_ss(line));
  }
}
#define check_cuda(call) _check_cuda(call, __FILE__, __LINE__)

int main() {
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Initialize vectors
    std::vector<float> h_A(N, 1.0f);
    std::vector<float> h_B(N, 2.0f);
    hipMemcpy(d_A, h_A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipStream_t stream;
    hipStreamCreate(&stream);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    hipEvent_t start, stop;
    check_cuda(hipEventCreate(&start));
    check_cuda(hipEventCreate(&stop));
    check_cuda(hipEventRecord(start, stream));
    
    for (int i = 0; i < NUM_GRAPHS; ++i) {
        for (int j = 0; j < NUM_OPERATIONS; ++j) {
            vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_A, d_B, d_C, N);
        }
        hipStreamSynchronize(stream);
    }
    
    check_cuda(hipEventRecord(stop, stream));
    check_cuda(hipEventSynchronize(stop));
    float msec = 0.0f;
    check_cuda(hipEventElapsedTime(&msec, start, stop));

    std::cout << "Time for 100 stream with hipEventRecord: " << msec << " milliseconds\n";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(stream);

    return 0;
}
